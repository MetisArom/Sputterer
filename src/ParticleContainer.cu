#include "hip/hip_runtime.h"
#include <random>

#include <thrust/distance.h>
#include <thrust/partition.h>

#include "ParticleContainer.cuh"
#include "cuda_helpers.cuh"
#include "gl_helpers.hpp"
#include "Constants.hpp"

// Setup RNG
__global__ void k_setup_rng (hiprandState *rng, uint64_t seed) {
  unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, tid, 0, &rng[tid]);
}

ParticleContainer::ParticleContainer (string name, size_t num, double mass, int charge)
  : name(std::move(name)), mass(mass), charge(charge) {

  // Allocate memory on GPU
  d_position.resize(num);
  d_velocity.resize(num);
  d_weight.resize(num);
  d_tmp.resize(num);
  d_rng.resize(num);

  // Set up RNG for later use
  size_t block_size = 512;
  k_setup_rng<<<num/block_size, block_size>>>(thrust::raw_pointer_cast(d_rng.data()), time(nullptr));
  std::cout << "GPU RNG state initialized." << std::endl;
}

void
ParticleContainer::add_particles (const host_vector<float3> &pos, const host_vector<float3> &vel
                                  , const host_vector<float> &w) {
  auto n = static_cast<int>(std::min({pos.size(), vel.size(), w.size()}));
  if (n == 0) return;

  position.resize(num_particles + n);
  velocity.resize(num_particles + n);
  weight.resize(num_particles + n);

  // Copy particles to CPU arrays
  for (size_t i = 0; i < n; i++) {
    auto id = num_particles + i;
    position[id] = pos[i];
    velocity[id] = vel[i];
    weight[id] = w[i];
  }

  // Copy particles to GPU
  thrust::copy(position.begin() + num_particles, position.end(), d_position.begin() + num_particles);
  thrust::copy(velocity.begin() + num_particles, velocity.end(), d_velocity.begin() + num_particles);
  thrust::copy(weight.begin() + num_particles, weight.end(), d_weight.begin() + num_particles);

  num_particles += n;
}

void ParticleContainer::copy_to_cpu () {
  position = host_vector<float3>(d_position.begin(), d_position.begin() + num_particles);
  velocity = host_vector<float3>(d_velocity.begin(), d_velocity.begin() + num_particles);
  weight = host_vector<float>(d_weight.begin(), d_weight.begin() + num_particles);
}

void ParticleContainer::set_buffers () {
  // enable buffer
  this->mesh.set_buffers();
  glGenBuffers(1, &this->buffer);
}

void ParticleContainer::draw () {

  // Bind vertex array
  auto vao = this->mesh.vao;
  GL_CHECK(glBindVertexArray(vao));

  // Send over model matrix data
  auto mat_vector_size = static_cast<GLsizei>(this->num_particles*sizeof(vec3));
  GL_CHECK(glBindBuffer(GL_ARRAY_BUFFER, this->buffer));
  GL_CHECK(glBufferData(GL_ARRAY_BUFFER, mat_vector_size, &position[0], GL_DYNAMIC_DRAW));

  // Set attribute pointers for translation
  GL_CHECK(glEnableVertexAttribArray(2));
  GL_CHECK(glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, sizeof(vec3), nullptr));
  GL_CHECK(glVertexAttribDivisor(2, 1));

  // Bind element array buffer
  GL_CHECK(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->mesh.ebo));

  // Draw meshes
  GL_CHECK(glDrawElementsInstanced(GL_TRIANGLES, static_cast<unsigned int>(3*this->mesh.num_triangles), GL_UNSIGNED_INT
                                   , nullptr, num_particles));

  // unbind buffers
  GL_CHECK(glBindVertexArray(0));
  GL_CHECK(glBindBuffer(GL_ARRAY_BUFFER, 0));
  GL_CHECK(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0));
}


__host__ __device__ float carbon_diffuse_prob (float cos_incident_angle, float incident_energy_ev) {
  // fit parameters
  constexpr auto angle_offset = 1.6823f;
  constexpr auto energy_offset = 65.6925f;
  constexpr auto energy_scale = 34.5302f;

  auto fac = (cos_incident_angle - angle_offset)*logf((incident_energy_ev + energy_offset)/energy_scale);
  auto diffuse_coeff = 0.003f + fac*fac;
  return diffuse_coeff;
}

__host__ __device__ float3 sample_diffuse (const Triangle &tri, const float3 norm, float thermal_speed) {
  // sample from a cosine distribution
#if defined(CUDA_ARCH)
  auto c_tan1 = hiprand_normal(&local_state);
  auto c_tan2 = hiprand_normal(&local_state);
  auto c_norm = abs(hiprand_normal(&local_state));
#else
  auto c_tan1 = rand_normal();
  auto c_tan2 = rand_normal();
  auto c_norm = abs(rand_normal());
#endif

  // get tangent vectors
  // TODO: may be worth pre-computing these?
  auto tan1 = normalize(tri.v1 - tri.v0);
  auto tan2 = cross(tan1, norm);

  // Compute new velocity vector
  auto vel_refl = thermal_speed*(c_norm*norm + c_tan1*tan1 + c_tan2*tan2);
  return vel_refl;
}

DeviceParticleContainer ParticleContainer::data () {
  DeviceParticleContainer pc;
  pc.position = thrust::raw_pointer_cast(this->d_position.data());
  pc.velocity = thrust::raw_pointer_cast(this->d_velocity.data());
  pc.weight = thrust::raw_pointer_cast(this->d_weight.data());
  pc.rng = thrust::raw_pointer_cast(this->d_rng.data());
  pc.num_particles = this->num_particles;
  return pc;
}

__global__ void
k_evolve (DeviceParticleContainer pc
          , const Triangle *tris, const size_t num_triangles
          , const Material *materials, const size_t *material_ids
          , int *collected, const float dt) {

  // Thread ID, i.e. what particle we're currently moving
  unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;

  using namespace constants;

  // Particle mass
  // FIXME: currently hard-coded to carbon, easy to fix by passing in mass as a param
  const double mass = 12.011*m_u;

  // Particle energy
  const double energy_factor = 0.5*mass/q_e;

  // k_B / m_u (for thermal speed calculations)
  const auto thermal_speed_factor = static_cast<float>(sqrt(k_b/mass));

  if (tid < pc.num_particles) {

    auto pos = pc.position[tid];
    auto vel = pc.velocity[tid];

    // Check for intersections with boundaries
    Ray ray{.origin = pos, .direction = dt*vel};
    auto closest_hit = ray.cast(tris, num_triangles);

    if (closest_hit.t <= 1) {
      auto &[_, t, hit_pos, norm, hit_triangle_id] = closest_hit;

      // Get material info where we hit
      auto &mat = materials[material_ids[hit_triangle_id]];

      // Generate a random number
      auto local_state = pc.rng[tid];
      auto uniform = hiprand_uniform(&local_state);

      // get incident angle and energy
      auto velnorm_2 = dot(vel, vel);
      auto cos_incident_angle = abs(dot(vel, -norm)/sqrt(velnorm_2));
      auto incident_energy_ev = static_cast<float>(energy_factor*velnorm_2);

      // Get sticking and diffuse coeff from model
      auto diffuse_coeff = carbon_diffuse_prob(cos_incident_angle, incident_energy_ev);
      auto sticking_coeff = 1.0f - diffuse_coeff;

      if (uniform < sticking_coeff) {
        // Particle sticks to surface
        pc.position[tid] = hit_pos;
        pc.velocity[tid] = float3(0.0f, 0.0f, 0.0f);

        // Record that we hit this triangle
        atomicAdd(&collected[hit_triangle_id], 1);

        // set weight negative to flag for removal
        // magnitude indicates which triangle we hit
        pc.weight[tid] = static_cast<float>(-hit_triangle_id);

      } else if (uniform < diffuse_coeff + sticking_coeff) {
        // Particle reflects diffusely based on surface temperature
        // TODO: pass thermal speed (or sqrt of temperature) instead of temperature to avoid this
        //
        auto sqrt_temp = sqrtf(mat.temperature_k);
        auto thermal_speed = thermal_speed_factor*sqrt_temp;
        auto vel_refl = sample_diffuse(tris[hit_triangle_id], norm, thermal_speed);

        // Get particle position
        // (assuming particle reflects ~instantaneously then travels according to new velocity vector)
        // TODO: most of this code is shared with below--worth unifying?
        auto final_pos = hit_pos + (1 - t)*dt*vel_refl;
        pc.position[tid] = final_pos;
        pc.velocity[tid] = vel_refl;
      } else {
        // Particle reflects specularly
        float3 vel_norm = dot(vel, norm)*norm;
        float3 vel_refl = vel - 2*vel_norm;

        auto final_pos = hit_pos + (1 - t)*dt*vel_refl;
        pc.position[tid] = final_pos;
        pc.velocity[tid] = vel_refl;
      }
    } else {
      pc.position[tid] = pos + dt*vel;
    }
  }
}

std::pair<dim3, dim3> ParticleContainer::get_kernel_launch_params (size_t block_size) const {
  auto grid_size = static_cast<int>(ceil(static_cast<float>(num_particles)/static_cast<float>(block_size)));
  dim3 grid(grid_size, 1, 1);
  dim3 block(block_size, 1, 1);
  return std::make_pair(grid, block);
}


void ParticleContainer::evolve (const float dt, const thrust::device_vector<Triangle> &tris
                                , const thrust::device_vector<Material> &mats, const thrust::device_vector<size_t> &ids
                                , thrust::device_vector<int> &collected) {


  // TODO: could move all of the device geometric info into a struct
  auto d_tri_ptr = thrust::raw_pointer_cast(tris.data());
  auto d_id_ptr = thrust::raw_pointer_cast(ids.data());
  auto d_mat_ptr = thrust::raw_pointer_cast(mats.data());

  auto d_col_ptr = thrust::raw_pointer_cast(collected.data());

  auto [grid, block] = get_kernel_launch_params();
  k_evolve<<<grid, block>>>(this->data(), d_tri_ptr, tris.size(), d_mat_ptr, d_id_ptr, d_col_ptr, dt);

  hipDeviceSynchronize();
}

float rand_uniform (float min, float max) {
  static std::default_random_engine rng;

  std::uniform_real_distribution<float> dist(min, max);
  return dist(rng);
}

float rand_normal (float mean, float std) {
  static std::default_random_engine rng;

  std::normal_distribution<float> dist(mean, std);
  return dist(rng);
}

void ParticleContainer::emit (Triangle &triangle, Emitter emitter, float dt) {
  auto num_emit = emitter.flux*triangle.area*dt;
  int num_emit_int = static_cast<int>(num_emit);
  auto remainder = num_emit - static_cast<float>(num_emit_int);

  auto u = rand_uniform();
  if (u < remainder) {
    num_emit_int += 1;
  }

  if (num_emit_int < 1) {
    return;
  }

  host_vector<float3> pos(num_emit_int);
  host_vector<float3> vel(num_emit_int);
  host_vector<float> w(num_emit_int, 1.0f);

  for (int i = 0; i < num_emit_int; i++) {
    auto pt = triangle.sample(rand_uniform(), rand_uniform());
    auto norm = emitter.reverse ? -triangle.norm : triangle.norm;
    // offset particle very slightly by norm
    auto tol = 0.0001f;
    pos[i] = pt + tol*norm;
    auto jitter = float3(
      rand_normal(0, emitter.spread), rand_normal(0, emitter.spread), rand_normal(0, emitter.spread));
    vel[i] = emitter.velocity*(norm + jitter);
  }

  add_particles(pos, vel, w);
}

__global__ void k_flag_oob (float3 *pos, float *weight, float radius2, float halflength, size_t n) {
  unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < n && weight[id] > 0) {
    auto r = pos[id];
    auto dist2 = r.x*r.x + r.y*r.y;
    if (dist2 > radius2 || r.z < -halflength || r.z > halflength) {
      // Particles that are oob get negative weight
      weight[id] = -1;
    }
  }
}

void ParticleContainer::flag_out_of_bounds (float radius, float length) {
  auto [grid, block] = get_kernel_launch_params();

  auto d_pos_ptr = thrust::raw_pointer_cast(d_position.data());
  auto d_wgt_ptr = thrust::raw_pointer_cast(d_weight.data());
  k_flag_oob<<<grid, block>>>(d_pos_ptr, d_wgt_ptr, radius*radius, length/2, num_particles);
  hipDeviceSynchronize();
}

struct IsPositive {
  __host__ __device__ bool operator() (const float &w) {
    return w > 0;
  }
};

void ParticleContainer::remove_flagged_particles () {
  // reorder positions and velocities so that particles with negative weight follow those with positive weight
  thrust::partition(d_position.begin(), d_position.begin() + num_particles, d_weight.begin(), IsPositive());
  thrust::partition(d_velocity.begin(), d_velocity.begin() + num_particles, d_weight.begin(), IsPositive());

  // reorder weights according to the same scheme as above
  // copy weights to temporary vector first
  // thrust partition likely is allocating some temporary memory
  // to avoid this, we would probably want to set up a custom allocator
  // c.f. https://github.com/NVIDIA/thrust/blob/1.6.0/examples/cuda/custom_temporary_allocation.cu
  // Alternatively, could use CUB device partition, which gives us more control to allocate temporary data
  // c.f. https://nvidia.github.io/cccl/cub/api/structcub_1_1DevicePartition.html#_CPPv4N3cub15DevicePartitionE
  thrust::copy(d_weight.begin(), d_weight.begin() + num_particles, d_tmp.begin());
  auto ret = thrust::partition(d_weight.begin(), d_weight.begin() + num_particles, d_tmp.begin(), IsPositive());

  // Reset number of particles to the middle of the partition
  num_particles = static_cast<int>(thrust::distance(d_weight.begin(), ret));
}


std::ostream &operator<< (std::ostream &os, ParticleContainer const &pc) {
  os << "==========================================================\n";
  os << "Particle container \"" << pc.name << "\"\n";
  os << "==========================================================\n";
  os << "Mass: " << pc.mass << "\n";
  os << "Charge: " << pc.charge << "\n";
  os << "Number of particles: " << pc.num_particles << "\n";
  os << "----------------------------------------------------------\n";
  os << "\tx\ty\tz\tvx\tvy\tvz\tw\t\n";
  os << "----------------------------------------------------------\n";
  for (int i = 0; i < pc.num_particles; i++) {
    os << "\t" << pc.position[i].x << " ";
    os << pc.position[i].y << "  ";
    os << pc.position[i].z << "  ";
    os << pc.velocity[i].x << "  ";
    os << pc.velocity[i].y << "  ";
    os << pc.velocity[i].z << "  ";
    os << pc.weight[i] << "\n";
  }
  os << "==========================================================\n";

  return os;
}