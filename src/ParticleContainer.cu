#include "hip/hip_runtime.h"
#include <random>

#include "ParticleContainer.cuh"
#include "cuda_helpers.cuh"

ParticleContainer::ParticleContainer(string name, double mass, int charge)
    : name(name)
    , mass(mass)
    , charge(charge) {}

void ParticleContainer::addParticles(vector<float> x, vector<float> y, vector<float> z, vector<float> ux,
                                     vector<float> uy, vector<float> uz, vector<float> w) {

    auto N = std::min({x.size(), y.size(), z.size(), ux.size(), uy.size(), uz.size(), w.size()});

    position.resize(numParticles + N);
    velocity.resize(numParticles + N);
    weight.resize(numParticles + N);

    // Add particles to CPU arrays
    for (int i = 0; i < N; i++) {
        position.at(i + numParticles) = {x.at(i), y.at(i), z.at(i)};
        velocity.at(i + numParticles) = {ux.at(i), uy.at(i), uz.at(i)};
        weight.at(i + numParticles)   = w.at(i);
    }

    // Copy particles to GPU
    // The starting memory address is numParticles * sizeof(float3)
    auto start_f3 = numParticles * sizeof(float3);
    auto size_f3  = N * sizeof(float3);
    auto start_f  = numParticles * sizeof(float);
    auto size_f   = N * sizeof(float);
    CUDA_CHECK(hipMemcpy(d_position.data() + start_f3, position.data() + start_f3, size_f3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_velocity.data() + start_f3, velocity.data() + start_f3, size_f3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weight.data() + start_f, weight.data() + start_f, size_f, hipMemcpyHostToDevice));

    numParticles += N;
}

void ParticleContainer::copyToCPU() {
    auto size_f3 = numParticles * sizeof(float3);
    auto size_f  = numParticles * sizeof(float);
    CUDA_CHECK(hipMemcpy(position.data(), d_position.data(), size_f3, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velocity.data(), d_velocity.data(), size_f3, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(weight.data(), d_weight.data(), size_f, hipMemcpyDeviceToHost));
}

#define MIN_T 100'000
#define TOL 1e-6

__host__ __device__ HitInfo hits_triangle (Ray ray, Triangle tri) {
    HitInfo info;

    // Find vectors for two edges sharing v1
    auto edge1 = tri.v1 - tri.v0;
    auto edge2 = tri.v2 - tri.v0;

    // Begin calculating determinant
    auto pvec = cross(ray.direction, edge2);
    auto det  = dot(edge1, pvec);

    // If determinant is near zero, ray lies in plane of triangle
    if (abs(det) < TOL) {
        return info;
    }

    // Calculate distance from v0 to ray origin
    auto tvec = ray.origin - tri.v0;

    // Calculate u parameter and test bounds
    auto u = dot(tvec, pvec) / det;
    if (u < 0.0 || u > 1.0) {
        return info;
    }

    auto qvec = cross(tvec, edge1);

    // Calculate v parameter and test bounds
    auto v = dot(ray.direction, qvec) / det;
    if (v < 0.0 || u + v > 1.0) {
        return info;
    }
    // Calculate t, ray intersects triangle
    auto t = dot(edge2, qvec) / det;

    info.hits = true;
    info.t    = t;

    if (dot(ray.direction, tri.norm) > 0) {
        info.norm = -tri.norm;
    } else {
        info.norm = tri.norm;
    }

    return info;
}

__global__ void k_push (float3 *position, float3 *velocity, int N, Triangle *tris, size_t numTriangles, float dt) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < N) {

        auto pos = position[id];
        auto vel = velocity[id];

        // Check for intersections with boundaries
        Ray ray{.origin = pos, .direction = dt * vel};

        HitInfo closest_hit{.hits = false, .t = static_cast<float>(MIN_T), .norm = {0.0, 0.0, 0.0}};
        HitInfo current_hit;
        for (size_t i = 0; i < numTriangles; i++) {
            current_hit = hits_triangle(ray, tris[i]);
            if (current_hit.hits && current_hit.t < closest_hit.t && current_hit.t >= 0) {
                closest_hit = current_hit;
            }
        }

        if (closest_hit.t <= 1) {
            auto &[_, t, norm] = closest_hit;

            float3 vel_norm = dot(vel, norm) * norm;
            float3 vel_refl = vel - 2 * vel_norm;

            auto hit_pos   = pos + t * dt * vel;
            auto final_pos = hit_pos + (1 - t) * dt * vel_refl;

            position[id] = final_pos;
            velocity[id] = vel_refl;

        } else {
            position[id] = pos + dt * vel;
        }
    }
}

void ParticleContainer::push(const float dt, const cuda::vector<Triangle> &tris) {
    const int BLOCK_SIZE = 32;
    const int GRID_SIZE  = static_cast<int>(ceil(static_cast<float>(numParticles) / BLOCK_SIZE));
    dim3      grid(GRID_SIZE, 1, 1);
    dim3      block(BLOCK_SIZE, 1, 1);

    k_push<<<grid, block>>>(d_position.data(), d_velocity.data(), numParticles, tris.data(), tris.size(), dt);

    hipDeviceSynchronize();
}

float randUniform (float min = 0.0f, float max = 1.0f) {
    static std::default_random_engine     rng;
    std::uniform_real_distribution<float> dist(min, max);
    return dist(rng);
}

void ParticleContainer::emit(Triangle &triangle, float flux, float dt) {
    auto numEmit    = flux * triangle.area * dt;
    int  intNumEmit = static_cast<int>(numEmit);
    auto remainder  = numEmit - intNumEmit;

    auto u = randUniform();
    if (u < remainder) {
        intNumEmit += 1;
    }
    // std::cout << "numEmit, intNumEmit, u, remainder: " << numEmit << ", " << intNumEmit << ", " << u << ", "
    //           << remainder << std::endl;

    float speed = 1.0;

    std::vector<float> x(intNumEmit, 0.0), y(intNumEmit, 0.0), z(intNumEmit, 0.0);
    std::vector<float> ux(intNumEmit, 0.0), uy(intNumEmit, 0.0), uz(intNumEmit, 0.0);
    std::vector<float> w(intNumEmit, 1.0);

    for (int i = 0; i < intNumEmit; i++) {
        auto pt  = triangle.sample(randUniform(), randUniform());
        x.at(i)  = pt.x;
        y.at(i)  = pt.y;
        z.at(i)  = pt.z;
        ux.at(i) = speed * triangle.norm.x;
        uy.at(i) = speed * triangle.norm.y;
        uz.at(i) = speed * triangle.norm.z;
    }

    addParticles(x, y, z, ux, uy, uz, w);
}

std::ostream &operator<< (std::ostream &os, ParticleContainer const &pc) {
    os << "==========================================================\n";
    os << "Particle container \"" << pc.name << "\"\n";
    os << "==========================================================\n";
    os << "Mass: " << pc.mass << "\n";
    os << "Charge: " << pc.charge << "\n";
    os << "Number of particles: " << pc.numParticles << "\n";
    os << "----------------------------------------------------------\n";
    os << "\tx\ty\tz\tvx\tvy\tvz\tw\t\n";
    os << "----------------------------------------------------------\n";
    for (int i = 0; i < pc.numParticles; i++) {
        os << "\t" << pc.position[i].x << "\t";
        os << pc.position[i].x << "\t";
        os << pc.position[i].x << "\t";
        os << pc.velocity[i].x << "\t";
        os << pc.velocity[i].x << "\t";
        os << pc.velocity[i].x << "\t";
        os << pc.weight[i] << "\t";
        os << "\n";
    }
    os << "==========================================================\n";

    return os;
}