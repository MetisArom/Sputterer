#include "hip/hip_runtime.h"
#include <random>

#include "ParticleContainer.cuh"
#include "cuda_helpers.cuh"

ParticleContainer::ParticleContainer(string name, double mass, int charge)
    : name(name)
    , mass(mass)
    , charge(charge) {

    // Allocate memory on GPU
    d_position.resize(MAX_PARTICLES);
    d_velocity.resize(MAX_PARTICLES);
    d_weight.resize(MAX_PARTICLES);
}

void ParticleContainer::addParticles(vector<float> x, vector<float> y, vector<float> z, vector<float> ux,
                                     vector<float> uy, vector<float> uz, vector<float> w) {

    auto N = std::min({x.size(), y.size(), z.size(), ux.size(), uy.size(), uz.size(), w.size()});

    // position.resize(numParticles + N);
    // velocity.resize(numParticles + N);
    // weight.resize(numParticles + N);

    // Add particles to CPU arrays
    for (int i = 0; i < N; i++) {
        position.push_back({x.at(i), y.at(i), z.at(i)});
        velocity.push_back({ux.at(i), uy.at(i), uz.at(i)});
        weight.push_back({w.at(i)});
    }

    // Copy particles to GPU
    // The starting memory address is numParticles

    thrust::copy(position.begin() + numParticles, position.end(), d_position.begin() + numParticles);
    thrust::copy(velocity.begin() + numParticles, velocity.end(), d_velocity.begin() + numParticles);
    thrust::copy(weight.begin() + numParticles, weight.end(), d_weight.begin() + numParticles);

    // CUDA_CHECK(hipMemcpy(d_position.data() + numParticles, position.data() + numParticles, N * sizeof(float3),
    //                       hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_velocity.data() + numParticles, velocity.data() + numParticles, N * sizeof(float3),
    //                       hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_weight.data() + numParticles, weight.data() + numParticles, N * sizeof(float3),
    //                       hipMemcpyHostToDevice));
    numParticles += N;
}

void ParticleContainer::copyToCPU() {
    // auto size_f3 = numParticles * sizeof(float3);
    // auto size_f  = numParticles * sizeof(float);
    // CUDA_CHECK(hipMemcpy(position.data(), d_position.data(), size_f3, hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpy(velocity.data(), d_velocity.data(), size_f3, hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpy(weight.data(), d_weight.data(), size_f, hipMemcpyDeviceToHost));

    thrust::copy(d_position.begin(), d_position.begin() + numParticles, position.begin());
    thrust::copy(d_velocity.begin(), d_velocity.begin() + numParticles, velocity.begin());
    thrust::copy(d_weight.begin(), d_weight.begin() + numParticles, weight.begin());
}

#define MIN_T 100'000
#define TOL 1e-6

__host__ __device__ HitInfo hits_triangle (Ray ray, Triangle tri) {
    HitInfo info;

    // Find vectors for two edges sharing v1
    auto edge1 = tri.v1 - tri.v0;
    auto edge2 = tri.v2 - tri.v0;

    // Begin calculating determinant
    auto pvec = cross(ray.direction, edge2);
    auto det  = dot(edge1, pvec);

    // If determinant is near zero, ray lies in plane of triangle
    if (abs(det) < TOL) {
        return info;
    }

    // Calculate distance from v0 to ray origin
    auto tvec = ray.origin - tri.v0;

    // Calculate u parameter and test bounds
    auto u = dot(tvec, pvec) / det;
    if (u < 0.0 || u > 1.0) {
        return info;
    }

    auto qvec = cross(tvec, edge1);

    // Calculate v parameter and test bounds
    auto v = dot(ray.direction, qvec) / det;
    if (v < 0.0 || u + v > 1.0) {
        return info;
    }
    // Calculate t, ray intersects triangle
    auto t = dot(edge2, qvec) / det;

    info.hits = true;
    info.t    = t;

    if (dot(ray.direction, tri.norm) > 0) {
        info.norm = -tri.norm;
    } else {
        info.norm = tri.norm;
    }

    return info;
}

__global__ void k_push (float3 *position, float3 *velocity, const int N, const Triangle *tris,
                        const size_t numTriangles, const float dt) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < N) {

        auto pos = position[id];
        auto vel = velocity[id];

        // Check for intersections with boundaries
        Ray ray{.origin = pos, .direction = dt * vel};

        HitInfo closest_hit{.hits = false, .t = static_cast<float>(MIN_T), .norm = {0.0, 0.0, 0.0}};
        HitInfo current_hit;
        for (size_t i = 0; i < numTriangles; i++) {
            current_hit = hits_triangle(ray, tris[i]);
            if (current_hit.hits && current_hit.t < closest_hit.t && current_hit.t >= 0) {
                closest_hit = current_hit;
            }
        }

        if (closest_hit.t <= 1) {
            auto &[_, t, norm] = closest_hit;

            float3 vel_norm = dot(vel, norm) * norm;
            float3 vel_refl = vel - 2 * vel_norm;

            auto hit_pos   = pos + t * dt * vel;
            auto final_pos = hit_pos + (1 - t) * dt * vel_refl;

            position[id] = final_pos;
            velocity[id] = vel_refl;

        } else {
            position[id] = pos + dt * vel;
        }
    }
}

void ParticleContainer::push(const float dt, const thrust::device_vector<Triangle> &tris) {
    const int BLOCK_SIZE = 32;
    const int GRID_SIZE  = static_cast<int>(ceil(static_cast<float>(numParticles) / BLOCK_SIZE));
    dim3      grid(GRID_SIZE, 1, 1);
    dim3      block(BLOCK_SIZE, 1, 1);

    auto d_pos_ptr = thrust::raw_pointer_cast(d_position.data());
    auto d_vel_ptr = thrust::raw_pointer_cast(d_velocity.data());
    auto d_tri_ptr = thrust::raw_pointer_cast(tris.data());

    k_push<<<grid, block>>>(d_pos_ptr, d_vel_ptr, numParticles, d_tri_ptr, tris.size(), dt);

    hipDeviceSynchronize();
}

float randUniform (float min = 0.0f, float max = 1.0f) {
    static std::default_random_engine     rng;
    std::uniform_real_distribution<float> dist(min, max);
    return dist(rng);
}

void ParticleContainer::emit(Triangle &triangle, float flux, float dt) {
    auto numEmit    = flux * triangle.area * dt;
    int  intNumEmit = static_cast<int>(numEmit);
    auto remainder  = numEmit - intNumEmit;

    auto u = randUniform();
    if (u < remainder) {
        intNumEmit += 1;
    }

    if (intNumEmit < 1) {
        return;
    }

    float speed = -1.0;

    std::vector<float> x(intNumEmit, 0.0), y(intNumEmit, 0.0), z(intNumEmit, 0.0);
    std::vector<float> ux(intNumEmit, 0.0), uy(intNumEmit, 0.0), uz(intNumEmit, 0.0);
    std::vector<float> w(intNumEmit, 1.0);

    for (int i = 0; i < intNumEmit; i++) {
        auto pt  = triangle.sample(randUniform(), randUniform());
        x.at(i)  = pt.x;
        y.at(i)  = pt.y;
        z.at(i)  = pt.z;
        ux.at(i) = speed * triangle.norm.x;
        uy.at(i) = speed * triangle.norm.y;
        uz.at(i) = speed * triangle.norm.z;
    }

    addParticles(x, y, z, ux, uy, uz, w);
}

std::ostream &operator<< (std::ostream &os, ParticleContainer const &pc) {
    os << "==========================================================\n";
    os << "Particle container \"" << pc.name << "\"\n";
    os << "==========================================================\n";
    os << "Mass: " << pc.mass << "\n";
    os << "Charge: " << pc.charge << "\n";
    os << "Number of particles: " << pc.numParticles << "\n";
    os << "----------------------------------------------------------\n";
    os << "\tx\ty\tz\tvx\tvy\tvz\tw\t\n";
    os << "----------------------------------------------------------\n";
    for (int i = 0; i < pc.numParticles; i++) {
        os << "\t" << pc.position[i].x << " ";
        os << pc.position[i].y << "  ";
        os << pc.position[i].z << "  ";
        os << pc.velocity[i].x << "  ";
        os << pc.velocity[i].y << "  ";
        os << pc.velocity[i].z << "  ";
        os << pc.weight[i] << "\n";
    }
    os << "==========================================================\n";

    return os;
}