#include "hip/hip_runtime.h"
// C++ headers
#include <iostream>
#include <string>
#include <vector>

// GLM headers
#include <glm/glm.hpp>

// ImGUI headers
#include "imgui.h"

// My headers (c++)
#include "app.hpp"
#include "input.hpp"
#include "mesh.hpp"
#include "shader.hpp"
#include "surface.hpp"
#include "window.hpp"

// My headers (CUDA)
#include "cuda.cuh"
#include "particle_container.cuh"
#include "triangle.cuh"

using std::vector, std::string;

string printTime (double time_s) {
    char   buf[64];
    int    factor = 1;
    string str    = "s";

    if (time_s < 1e-6) {
        factor = 1'000'000'000;
        str    = "ns";
    } else if (time_s < 1e-3) {
        factor = 1'000'000;
        str    = "us";
    } else if (time_s < 1) {
        factor = 1000;
        str    = "ms";
    }

    sprintf(buf, "%.3f %s", time_s * factor, str.c_str());

    return {buf};
}

int main (int argc, char *argv[]) {
    // Handle command line arguments
    string filename("../input.toml");
    if (argc > 1) {
        filename = argv[1];
    }

    Window window("Sputterer", app::SCR_WIDTH, app::SCR_HEIGHT);

    glfwSetFramebufferSizeCallback(window.window, app::framebufferSizeCallback);
    glfwSetCursorPosCallback(window.window, app::mouseCursorCallback);
    glfwSetScrollCallback(window.window, app::scrollCallback);

    Shader shader("../shaders/shader.vert", "../shaders/shader.frag");
    shader.use();

    Input input(filename);
    input.read();

    std::cout << "Input read." << std::endl;

    app::camera.orientation = glm::normalize(glm::vec3(input.chamberRadius));
    app::camera.distance    = 2.0f * input.chamberRadius;
    app::camera.yaw         = -135;
    app::camera.pitch       = 30;
    app::camera.updateVectors();

    // Create particle container, including any explicitly-specified initial particles
    ParticleContainer pc{"noname", 1.0f, 1};
    pc.addParticles(input.particle_x, input.particle_y, input.particle_z, input.particle_vx, input.particle_vy,
                    input.particle_vz, input.particle_w);

    vec3 particleColor{0.05f};
    vec3 particleColorOOB{1.0f, 0.2f, 0.2f};
    vec3 particleScale{0.01f};

    // Read mesh from file
    Mesh particleMesh{};
    particleMesh.readFromObj("../o_sphere.obj");
    particleMesh.setBuffers();

    // construct triangles
    host_vector<Triangle> h_triangles;
    host_vector<size_t>   h_materialIDs;
    host_vector<Material> h_materials;
    host_vector<char>     h_to_collect;
    std::vector<int>      collect_inds;
    std::vector<string>   surfaceNames;

    for (size_t id = 0; id < input.surfaces.size(); id++) {
        const auto &surf     = input.surfaces.at(id);
        const auto &mesh     = surf.mesh;
        const auto &material = surf.material;

        surfaceNames.push_back(surf.name);
        h_materials.push_back(surf.material);

        for (const auto &[i1, i2, i3] : mesh.triangles) {
            auto model = surf.transform.getMatrix();
            auto v1    = make_float3(model * glm::vec4(mesh.vertices[i1].pos, 1.0));
            auto v2    = make_float3(model * glm::vec4(mesh.vertices[i2].pos, 1.0));
            auto v3    = make_float3(model * glm::vec4(mesh.vertices[i3].pos, 1.0));

            h_triangles.push_back({v1, v2, v3});
            h_materialIDs.push_back(id);
            if (material.collect) {
                collect_inds.push_back(static_cast<int>(h_triangles.size()) - 1);
            }
        }
    }

    host_vector<int> collected(collect_inds.size(), 0);

    std::cout << "Meshes read." << std::endl;

    // Send mesh data to GPU. Really slow for some reason (multiple seconds)!
    device_vector<Triangle> d_triangles{h_triangles};
    device_vector<size_t>   d_surfaceIDs{h_materialIDs};
    device_vector<Material> d_materials{h_materials};
    device_vector<int>      d_collected(h_triangles.size(), 0);

    std::cout << "Mesh data sent to GPU." << std::endl;

    // Create timing objects
    size_t frame = 0;

    float  avgTimeCompute = 0.0f, avgTimeTotal = 0.0f;
    float  iterReset    = 100;
    float  timeConst    = 1 / iterReset;
    double physicalTime = 0, physicalTimestep = 0;

    cuda::event start{}, stopCompute{}, stopCopy{};

    std::cout << "Beginning main loop." << std::endl;

    while (window.open) {

        Window::beginRenderLoop();

        // Timing info
        auto flags = ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoResize |
                     ImGuiWindowFlags_NoInputs | ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoSavedSettings;
        float  padding      = 0.0f;
        ImVec2 bottom_right = ImVec2(ImGui::GetIO().DisplaySize.x - padding, ImGui::GetIO().DisplaySize.y - padding);
        ImGui::SetNextWindowPos(bottom_right, ImGuiCond_Always, ImVec2(1.0, 1.0));
        ImGui::Begin("Frame time", nullptr, flags);
        ImGui::Text("Simulation timestep: %s\nSimulation time: %s\nCompute time: %.3f ms (%.2f%% data transfer)  "
                    "\nParticles: %i",
                    printTime(physicalTimestep).c_str(), printTime(physicalTime).c_str(), avgTimeCompute,
                    (1.0f - avgTimeCompute / avgTimeTotal) * 100, pc.numParticles);
        ImGui::End();

        // Table of collected particle amounts
        auto   tableFlags  = ImGuiTableFlags_BordersH;
        ImVec2 bottom_left = ImVec2(0, ImGui::GetIO().DisplaySize.y - padding);
        ImGui::SetNextWindowPos(bottom_left, ImGuiCond_Always, ImVec2(0.0, 1.0));
        ImGui::Begin("Particle collection info", nullptr, flags);
        if (ImGui::BeginTable("Table", 3, tableFlags)) {
            ImGui::TableNextRow();
            ImGui::TableNextColumn();
            ImGui::Text("Surface name");
            ImGui::TableNextColumn();
            ImGui::Text("Triangle ID");
            ImGui::TableNextColumn();
            ImGui::Text("Collection rate (#/s)");
            for (int row = 0; row < collect_inds.size(); row++) {
                auto triangleID = collect_inds[row];
                ImGui::TableNextRow();
                ImGui::TableNextColumn();
                ImGui::Text("%s", surfaceNames.at(h_materialIDs[triangleID]).c_str());
                ImGui::TableNextColumn();
                ImGui::Text("%i", static_cast<int>(triangleID));
                ImGui::TableNextColumn();
                ImGui::Text("%.3e", static_cast<double>(collected[row]) / physicalTime);
            }
            ImGui::EndTable();
        }
        ImGui::End();

        // frame timing for rendering
        auto currentFrame = static_cast<float>(glfwGetTime());
        app::deltaTime    = currentFrame - app::lastFrame;
        app::lastFrame    = currentFrame;
        app::processInput(window.window);

        auto thisTimestep = input.timestep * app::deltaTime;
        physicalTime += thisTimestep;
        physicalTimestep = (1 - timeConst) * physicalTimestep + timeConst * input.timestep * app::deltaTime;

        // record compute start time
        if (frame > 1) {
            start.record();

            // Emit particles
            size_t triCount{0};
            for (const auto &surf : input.surfaces) {
                auto &emitter = surf.emitter;
                if (!emitter.emit) {
                    continue;
                }

                for (size_t i = 0; i < surf.mesh.numTriangles; i++) {
                    pc.emit(h_triangles[i], emitter, physicalTimestep);
                }
                triCount += surf.mesh.numTriangles;
            }

            // Push particles
            pc.push(physicalTimestep, d_triangles, d_surfaceIDs, d_materials, d_collected);

            // Remove particles that are out of bounds
            pc.flagOutOfBounds(input.chamberRadius, input.chamberLength);
            pc.removeFlaggedParticles();
            stopCompute.record();

            // Track particles collected by each triangle flagged 'collect'
            for (int id = 0; id < collect_inds.size(); id++) {
                auto oldVal  = collected[id];
                auto d_begin = d_collected.begin() + collect_inds[id];
                thrust::copy(d_begin, d_begin + 1, collected.begin() + id);
                collected[id] += oldVal;
            }

            // Copy particle data back to CPU
            pc.copyToCPU();

            stopCopy.record();

            float elapsedCompute, elapsedCopy;
            elapsedCompute = cuda::eventElapsedTime(start, stopCompute);
            elapsedCopy    = cuda::eventElapsedTime(start, stopCopy);

            avgTimeCompute = (1 - timeConst) * avgTimeCompute + timeConst * elapsedCompute;
            avgTimeTotal   = (1 - timeConst) * avgTimeTotal + timeConst * elapsedCopy;
        }

        // update camera projection
        shader.updateView(app::camera, app::aspectRatio);

        for (const auto &surface : input.surfaces) {
            // set the model matrix
            surface.mesh.draw(shader, surface.transform, surface.color);
        }

        for (int i = 0; i < pc.numParticles; i++) {
            // this is pretty inefficient, as we have to copy a lot of identical vertex and normal data over to the
            // GPU for each particle Ideally, we'd use instancing to do better, and only transfer the model matrix
            // over at each timestep see https://learnopengl.com/Advanced-OpenGL/Instancing
            Transform t;
            t.scale     = particleScale;
            t.translate = glm::vec3{pc.position[i].x, pc.position[i].y, pc.position[i].z};
            auto color  = pc.weight[i] > 0 ? particleColor : particleColorOOB;
            particleMesh.draw(shader, t, color);
        }

        window.endRenderLoop();
        frame += 1;
    }

    std::cout << "Program terminated successfully." << std::endl;

    return 0;
}
